#include "hip/hip_runtime.h"
#include "object.h"

#ifndef MACROCELL_SIZE_MIP
#error MACROCELL_SIZE_MIP is not defined in the CMakeLists.txt
#endif
__device__ constexpr int MACROCELL_SIZE = 1 << MACROCELL_SIZE_MIP;

namespace vnr {

__forceinline__ __device__ void 
update_single_macrocell(const vec3i& voxel, const vec3i& dims, float* __restrict__ macrocells, float value)
{
  const vec3i cell = {
    voxel.x >> MACROCELL_SIZE_MIP,
    voxel.y >> MACROCELL_SIZE_MIP,
    voxel.z >> MACROCELL_SIZE_MIP,
  };

  if (cell.x < 0 || cell.x >= dims.x) return;
  if (cell.y < 0 || cell.y >= dims.y) return;
  if (cell.z < 0 || cell.z >= dims.z) return;

  assert(cell.x < dims.x);
  assert(cell.y < dims.y);
  assert(cell.z < dims.z);
  assert(cell.x >= 0);
  assert(cell.y >= 0);
  assert(cell.z >= 0);

  const uint32_t idx = cell.x + cell.y * dims.x + cell.z * dims.y * dims.x;
  float* __restrict__ vmin = macrocells + 2 * idx;
  float* __restrict__ vmax = macrocells + 2 * idx + 1;

  // All the value ranges are initialized as zero. Because all the values are within
  // range [0, 1], we can still compute a global min/max by adding a -1/+1 offset. 
  // We need to remove this offset when accessing value ranges.
  atomicMin(vmin, value - 1.f);
  atomicMax(vmax, value + 1.f);
}

__global__ void
update_macrocell_explicit(const uint32_t n_elements,
                          const vec3f* __restrict__ coords,
                          const float* __restrict__ values,
                          const vec3i dims,
                          const vec3i macrocell_dims,
                          float* __restrict__ macrocells)
{
  const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n_elements)
    return;

  const vec3f coord = coords[i];
  const float value = values[i];

  const uint32_t x = clamp((uint32_t)floorf(coord.x * dims.x), uint32_t(0), uint32_t(dims.x-1));
  const uint32_t y = clamp((uint32_t)floorf(coord.y * dims.y), uint32_t(0), uint32_t(dims.y-1));
  const uint32_t z = clamp((uint32_t)floorf(coord.z * dims.z), uint32_t(0), uint32_t(dims.z-1));

  const int sx = (x % MACROCELL_SIZE) == 0 ? -1 : (x % MACROCELL_SIZE) == (MACROCELL_SIZE-1) ? 1 : 0;
  const int sy = (y % MACROCELL_SIZE) == 0 ? -1 : (y % MACROCELL_SIZE) == (MACROCELL_SIZE-1) ? 1 : 0;
  const int sz = (z % MACROCELL_SIZE) == 0 ? -1 : (z % MACROCELL_SIZE) == (MACROCELL_SIZE-1) ? 1 : 0;

  update_single_macrocell(vec3i(x,      y,      z     ), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x + sx, y,      z     ), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x,      y + sy, z     ), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x + sx, y + sy, z     ), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x,      y,      z + sz), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x + sx, y,      z + sz), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x,      y + sy, z + sz), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x + sx, y + sy, z + sz), macrocell_dims, macrocells, value);
}

__global__ void
update_macrocell_implicit(const uint32_t n_elements,
                          const uint64_t n_offset,
                          const vec3i dims,
                          const hipTextureObject_t texture,
                          const vec3i macrocell_dims,
                          float* __restrict__ macrocells)
{
  const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n_elements)
    return;

  const uint64_t idx = i + n_offset;
  const uint64_t stride = (uint64_t)dims.x * dims.y;

  const uint32_t x =  idx % dims.x;
  const uint32_t y = (idx % stride) / dims.x;
  const uint32_t z =  idx / stride;

  const float fx = (x + 0.5f) / dims.x;
  const float fy = (y + 0.5f) / dims.y;
  const float fz = (z + 0.5f) / dims.z;
  const float value = tex3D<float>(texture, fx, fy, fz);

  const int sx = (x % MACROCELL_SIZE) == 0 ? -1 : (x % MACROCELL_SIZE) == (MACROCELL_SIZE-1) ? 1 : 0;
  const int sy = (y % MACROCELL_SIZE) == 0 ? -1 : (y % MACROCELL_SIZE) == (MACROCELL_SIZE-1) ? 1 : 0;
  const int sz = (z % MACROCELL_SIZE) == 0 ? -1 : (z % MACROCELL_SIZE) == (MACROCELL_SIZE-1) ? 1 : 0;

  update_single_macrocell(vec3i(x,      y,      z     ), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x + sx, y,      z     ), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x,      y + sy, z     ), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x + sx, y + sy, z     ), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x,      y,      z + sz), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x + sx, y,      z + sz), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x,      y + sy, z + sz), macrocell_dims, macrocells, value);
  update_single_macrocell(vec3i(x + sx, y + sy, z + sz), macrocell_dims, macrocells, value);
}

__global__ void // computing macrocell value range offline
macrocell_value_range_kernel(const uint32_t mcDimsX,
                             const uint32_t mcDimsY,
                             const uint32_t mcDimsZ,
                             const uint32_t mcWidth,
                             vec2f* __restrict__ mcData,
                             const vec3i volumeDims,
                             hipTextureObject_t volumeTexture)
{
  // 3D kernel launch
  vec3i mcID(threadIdx.x+blockIdx.x*blockDim.x,
             threadIdx.y+blockIdx.y*blockDim.y,
             threadIdx.z+blockIdx.z*blockDim.z);

  if (mcID.x >= mcDimsX) return;
  if (mcID.y >= mcDimsY) return;
  if (mcID.z >= mcDimsZ) return;

  int mcIdx = mcID.x + mcDimsX*(mcID.y + mcDimsY*mcID.z);
  vec2f &mc = mcData[mcIdx];

  // compute begin/end of VOXELS for this macro-cell
  vec3i begin = max(mcID  * vec3i(mcWidth) - 1, vec3i(0));
  vec3i end   = min(begin + vec3i(mcWidth) + /* plus one for tri-lerp!*/ 2, volumeDims);

  range1f valueRange;
  for (int iz = begin.z; iz < end.z; iz++)
    for (int iy = begin.y; iy < end.y; iy++)
      for (int ix = begin.x; ix < end.x; ix++) {
          float f;
          tex3D(&f, volumeTexture, 
                (ix + 0.5f) / volumeDims.x, 
                (iy + 0.5f) / volumeDims.y, 
                (iz + 0.5f) / volumeDims.z);
          valueRange.extend(f);
        }
  mc.x = valueRange.lo - 1.f;
  mc.y = valueRange.hi + 1.f;
}

__global__ void // compute macrocell opacity all together
macrocell_max_opacity_kernel(const uint32_t num_cells, const DeviceTransferFunction tfn, 
                             const vec2f* __restrict__ cell_value_range, 
                             float* __restrict__ cell_max_opacity)
{
  extern __shared__ float shared_alphas[];

  const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
  assert(blockDim.x == tfn.alphas.length);

  // load tfn into shared memory (assume the number of threads per group equals the length of the alpha array)
  shared_alphas[threadIdx.x] = ((float*)tfn.alphas.rawptr)[threadIdx.x];
  __syncthreads();
  const float* __restrict__ alphas = shared_alphas;

  // access macrocell value range
  if (i >= num_cells) return;
  auto range = cell_value_range[i];
  range.x += 1.f;
  range.y -= 1.f; // see function: update_single_macrocell

  // compute the max opacity for the cell
  assert(tfn.alphas.length > 0); // for the first frame, tfn.alphas.length might be zero

  const auto lower = (clamp(range.x, tfn.range.lower, tfn.range.upper) - tfn.range.lower) * tfn.range_rcp_norm;
  const auto upper = (clamp(range.y, tfn.range.lower, tfn.range.upper) - tfn.range.lower) * tfn.range_rcp_norm;
  uint32_t i_lower = floorf(fmaf(lower, float(tfn.alphas.length-1), 0.5f)) - 1;
  uint32_t i_upper = floorf(fmaf(upper, float(tfn.alphas.length-1), 0.5f)) + 1;
  i_lower = clamp<uint32_t>(i_lower, 0, tfn.alphas.length-1);
  i_upper = clamp<uint32_t>(i_upper, 0, tfn.alphas.length-1);

  assert(i_lower < tfn.alphas.length);
  assert(i_upper < tfn.alphas.length);

  float opacity = 0.f;
  for (auto i = i_lower; i <= i_upper; ++i) {
    opacity = std::max(opacity, alphas[i]);
  }
  cell_max_opacity[i] = opacity;
}

void MacroCell::set_shape(vec3i dims)
{
  m_volume_dims = dims;

  m_dims = util::div_round_up(dims, vec3i(MACROCELL_SIZE));
  m_spacings = vec3f(MACROCELL_SIZE) / vec3f(dims);
}

void MacroCell::set_external(MacroCell& external)
{
  m_volume_dims = external.m_volume_dims;
  m_dims = external.m_dims;
  m_spacings = external.m_spacings;
  m_max_opacity_buffer.set_external(external.m_max_opacity_buffer);
  m_value_range_buffer.set_external(external.m_value_range_buffer);
  m_is_external = true;
}

void MacroCell::allocate()
{
  m_value_range_buffer.resize(m_dims.long_product() * sizeof(range1f), nullptr);
  m_value_range_buffer.memset(0, nullptr);
  m_max_opacity_buffer.resize(m_dims.long_product() * sizeof(float), nullptr);
  m_is_external = false;
}

void MacroCell::compute_everything(hipTextureObject_t volume)
{
  for (int z = 0; z < m_volume_dims.z; ++z) {
    util::linear_kernel(update_macrocell_implicit, 0, 0, 
                        (uint32_t)m_volume_dims.x * m_volume_dims.y, 
                        (uint64_t)m_volume_dims.x * m_volume_dims.y * z, 
                        m_volume_dims, volume, m_dims,
                        (float*)d_value_range());
  }
  // util::trilinear_kernel(macrocell_value_range_kernel, 0, 0, 
  //     m_dims.x, m_dims.y, m_dims.z, 
  //     MACROCELL_SIZE, (vec2f*)d_value_range(), 
  //     m_volume_dims, volume);
}

void MacroCell::update_explicit(vec3f* d_coords, float* d_values, size_t count, hipStream_t stream)
{
  util::linear_kernel(update_macrocell_explicit, 0, stream, count, 
                      d_coords, d_values, m_volume_dims, m_dims, 
                      (float*)d_value_range());
}

void MacroCell::update_max_opacity(const DeviceTransferFunction& tfn, hipStream_t stream) 
{
  if (tfn.alphas.length <= 0) return;

  const size_t shmem = tfn.alphas.length * sizeof(float);
  const size_t n_elements = m_dims.long_product();

  macrocell_max_opacity_kernel<<<util::div_round_up(n_elements, tfn.alphas.length), tfn.alphas.length, shmem, stream>>>(
    n_elements, tfn, (vec2f*)d_value_range(), (float*)d_max_opacity()
  );
}

void
TransferFunctionObject::clean()
{
  if (tfn_color_array_handler) {
    CUDA_CHECK_NOEXCEPT(hipFreeArray(tfn_color_array_handler));
    tfn_color_array_handler = NULL;
    util::total_n_bytes_allocated() -= tfn.colors.length * sizeof(float4);
#ifdef VNR_VERBOSE_MEMORY_ALLOCS
    printf("[mem] Array1D free %s\n", util::prettyBytes(tfn.colors.length * sizeof(float4)).c_str());
#endif
  }
  if (tfn.colors.data) {
    CUDA_CHECK_NOEXCEPT(hipDestroyTextureObject(tfn.colors.data));
    tfn.colors.data = { 0 };
  }
  if (tfn.colors.rawptr) {
    CUDA_CHECK_NOEXCEPT(hipFree(tfn.colors.rawptr));
    tfn.colors.rawptr = nullptr;
    util::total_n_bytes_allocated() -= tfn.colors.length * sizeof(float4);
#ifdef VNR_VERBOSE_MEMORY_ALLOCS
    printf("[mem] Linear free %s\n", util::prettyBytes(tfn.colors.length * sizeof(float4)).c_str());
#endif
  }
  tfn.colors.length = 0;

  if (tfn_alpha_array_handler) {
    CUDA_CHECK_NOEXCEPT(hipFreeArray(tfn_alpha_array_handler));
    tfn_color_array_handler = NULL;
    util::total_n_bytes_allocated() -= tfn.alphas.length * sizeof(float);
#ifdef VNR_VERBOSE_MEMORY_ALLOCS
    printf("[mem] Array1D free %s\n", util::prettyBytes(tfn.alphas.length * sizeof(float)).c_str());
#endif
  }
  if (tfn.alphas.data) {
    CUDA_CHECK_NOEXCEPT(hipDestroyTextureObject(tfn.alphas.data));
    tfn.alphas.data = { 0 };
  }
  if (tfn.alphas.rawptr) {
    CUDA_CHECK_NOEXCEPT(hipFree(tfn.alphas.rawptr));
    tfn.alphas.rawptr = nullptr;
    util::total_n_bytes_allocated() -= tfn.alphas.length * sizeof(float);
#ifdef VNR_VERBOSE_MEMORY_ALLOCS
    printf("[mem] Linear free %s\n", util::prettyBytes(tfn.alphas.length * sizeof(float)).c_str());
#endif
  }
  tfn.alphas.length = 0;
}

void 
TransferFunctionObject::set_transfer_function(const std::vector<vec3f>& c, const std::vector<vec2f>& o, const range1f& r, hipStream_t stream)
{
  std::vector<float4> colors_data;
  std::vector<float> alphas_data;
  colors_data.resize(c.size());
  for (int i = 0; i < colors_data.size(); ++i) {
    colors_data[i].x = c[i].x;
    colors_data[i].y = c[i].y;
    colors_data[i].z = c[i].z;
    colors_data[i].w = 1.f;
  }
  alphas_data.resize(o.size());
  for (int i = 0; i < alphas_data.size(); ++i) {
    alphas_data[i] = o[i].y;
  }

  TRACE_CUDA;

  tfn.range = r;
  tfn.range_rcp_norm = 1.f / tfn.range.span();

  TRACE_CUDA;

  if (!colors_data.empty())
    CreateArray1DFloat4(stream, colors_data, tfn_color_array_handler, tfn.colors);
  
  TRACE_CUDA;

  if (!alphas_data.empty())
    CreateArray1DScalar(stream, alphas_data, tfn_alpha_array_handler, tfn.alphas);

  TRACE_CUDA;
}

void
SimpleVolume::load(const MultiVolume& descriptor, std::string sampling_mode, bool save_volume)
{
  desc = descriptor;
  mode = sampling_mode;
  sampler.load(desc, sampling_mode, save_volume);
  tex = sampler.texture();
  if (tex) {
    macrocell.set_shape(desc.dims);
    macrocell.allocate();
    macrocell.compute_everything(tex);
  }
}

void 
SimpleVolume::set_current_timestep(int index) 
{ 
  sampler.set_current_volume_index(index); 
  if (tex && !macrocell.is_external()) {
    macrocell.compute_everything(tex);
  }
}

void SimpleVolume::set_transfer_function(const std::vector<vec3f>& c, const std::vector<vec2f>& o, const range1f& r)
{
  tfn.set_transfer_function(c, o, r, nullptr);
  if (macrocell.allocated()) {
    macrocell.update_max_opacity(tfn.tfn, nullptr);
  }
}

void SimpleVolume::set_data_transform(affine3f transform)
{
  sampler.set_transform(transform);
}

}
